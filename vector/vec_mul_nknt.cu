/*
	A CUDA code, initalizes 2 float vectors, and multiply them using GPU, stores the result in 3rd vector.
	The operations done on N Kernal, N thread each
*/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <malloc.h>
#define THREADS_PER_BLOCK 512
float *a,*b,*c;
float *da,*db,*dc;
int *dn;
int size;

//Function allocates a float vector, and return its address
float *allocateVector(int size) {
	float *t;
	t=(float *)malloc(sizeof(float) * size);
	return t;
}

//Function takes a float vector, and deallocate it
float *deallocateVector(float *v) {
	free(v);
	return v;
}

//Function takes a vector and its size, fill it with values
float *fillRand(float *v,int size) {
	int i;
	for(i=0;i<size;i++)
		v[i]=i+1;
	return v;
}

//Print a vector
void printVector(float *v,int s) {
	int i;
	printf("\n");
	for(i=0;i<s;i++) 
		printf("%2.3f ",v[i]);
}

//The code performs vector multiplication, in each block 
__global__ void dotproduct(float *da,float *db,float *dc,int *dn) {
	/*
		threadIdx.x	: Thread number in current block
		blockIdx.x	: Block number
		blockDim.x	: Size of the block
	*/
	int loc=threadIdx.x + blockIdx.x * blockDim.x;
	if ( loc < *dn ) 
		dc[loc] = da[loc] * db[loc];
}

int main(int argc,char **argv) {
	//commStart, commStop used to record time 
	hipEvent_t commStart,commStop;
	float comptime;
	//Check for passed parameters, the program needs one parameter , the vector size
	if ( argc != 2 ) {
		printf("\nNot enough parameters");
		return 1;
	}
	//Reads the parameters and convert them from array of characters to integer
	size=atoi(argv[1]);
	//printf("\nVector of size  = %d ", size);
	//printf("\nAllocating vectors");
	//Allocation 3 vectors in host memory
	a=allocateVector(size);
	b=allocateVector(size);
	c=allocateVector(size);

	//Allocation of 3 vectors in device memory
	hipMalloc((void **)&da,size*sizeof(float));
	hipMalloc((void **)&db,size*sizeof(float));
	hipMalloc((void **)&dc,size*sizeof(float));
	hipMalloc((void **)&dn,sizeof(int));

	//Filling 2 host vectors
	//printf("\nFIllin in vector");
	a=fillRand(a,size);
	b=fillRand(b,size);

	hipEventCreate(&commStart);
	hipEventCreate(&commStop);
	hipEventRecord(commStart,0);

	//Copy content of vector a, and b to the vectors stored in GPU memory
	hipMemcpy(da,a,size*sizeof(float),hipMemcpyHostToDevice);
	hipMemcpy(db,b,size*sizeof(float),hipMemcpyHostToDevice);
	hipMemcpy(dn,&size,sizeof(int),hipMemcpyHostToDevice);


	//Launch the kernel with blocks = size, and block size =1
	dotproduct<<<size/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(da,db,dc,dn);


	//Copy back result from GPU memory vector dc, to the host memory vector c

	hipMemcpy(c,dc,size*sizeof(float),hipMemcpyDeviceToHost);

	hipEventRecord(commStop,0);

        hipEventSynchronize(commStop);
        hipEventElapsedTime(&comptime,commStart,commStop);
	comptime=comptime/1000;
	
	printf("\ndot product for size %d took %f seconds\n",size,comptime);

/*
	printf("\nIn");
	printVector(in,size);
	printf("\nOut");
	printVector(out,size);
	printf("\nDeallocating vectors");
*/
	a=deallocateVector(a);
	b=deallocateVector(b);
	c=deallocateVector(c);
	hipFree(da);
	hipFree(db);
	hipFree(dc);
	hipFree(dn);
	return 0;
}
